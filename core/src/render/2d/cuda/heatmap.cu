#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "render/utils/color/color_gradient.h"
#include "render/2d/heatmap.h"

namespace zilliz {
namespace render {

template<typename T>
__global__ void SetCountValue_gpu(float *out,
                                  uint32_t *in_x,
                                  uint32_t *in_y,
                                  T *in_c,
                                  int64_t num,
                                  int64_t width,
                                  int64_t height) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    for (; i < num; i += blockDim.x * gridDim.x) {
        uint32_t vertice_x = in_x[i];
        uint32_t vertice_y = in_y[i];
        int64_t index = vertice_y * width + vertice_x;
        if (index >= width * height)
            continue;
        out[index] += in_c[i];
    }
}

__global__ void
HeatMapArray_gpu(float *in_count, float *out_count, float *kernel, int64_t kernel_size, int64_t width, int64_t height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int count_index = y * width + x;
    if (in_count[count_index] > 1e-8) {
        int r = kernel_size / 2;
        for (int m = -r; m <= r; m++) {
            if (x + m < 0 || x + m >= width)
                continue;
            for (int n = -r; n <= r; n++) {
                if (y + n < 0 || y + n >= height)
                    continue;
                int kernel_index = (r + n) * (2 * r + 1) + (m + r);
                int dev_index = (y + n) * width + (x + m);
                out_count[dev_index] += in_count[count_index] * kernel[kernel_index];
            }
        }
    }
}

__global__ void
MeanKernel_gpu(float *img_in, float *img_out, int64_t r, int64_t img_w, int64_t img_h) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    double gradient = 0.0;
    if (r > 10) r = 10;
    int count = 0;
    if ((row >= 0) && (row < img_h) && (col >= 0) && (col < img_w)) {
        for (int m = -r; m <= r; m++) {
            if (row + m < 0 || row + m >= img_h)
                continue;
            for (int n = -r; n <= r; n++) {
                if (col + n < 0 || col + n >= img_w)
                    continue;
                int y = row + m;
                int x = col + n;
                gradient += img_in[y * img_w + x];
                count++;
            }
        }
        img_out[row * img_w + col] = gradient / count;
    }
}

template<typename T>
void HeatMap<T>::set_colors_gpu() {
    WindowParams window_params = heatmap_vega_.window_params();
    int64_t width = window_params.width();
    int64_t height = window_params.height();
    int64_t window_size = width * height;

    float *pix_count;
    hipMalloc((void **) &pix_count, window_size * sizeof(float));
    hipMemset(pix_count, 0, window_size * sizeof(float));
    SetCountValue_gpu < T > << <256, 1024 >>
        > (pix_count, vertices_x_.get(), vertices_y_.get(), count_.get(), num_vertices_, width, height);

    double scale = heatmap_vega_.map_scale() * 0.4;
    int d = pow(2, scale);
    int64_t kernel_size = d * 2 + 3;

    float *kernel = (float *) malloc(kernel_size * kernel_size * sizeof(float));
    guassiankernel2d(kernel, kernel_size, kernel_size, kernel_size, kernel_size);
    float *dev_kernel;
    hipMalloc((void **) &dev_kernel, kernel_size * kernel_size * sizeof(float));
    hipMemcpy(dev_kernel, kernel, kernel_size * kernel_size * sizeof(float), hipMemcpyHostToDevice);
    float *dev_count;
    hipMalloc((void **) &dev_count, window_size * sizeof(float));
    hipMemset(dev_count, 0, window_size * sizeof(float));

    const unsigned int blockW = 32;
    const unsigned int blockH = 32;
    const dim3 threadBlock(blockW, blockH);
    const dim3 grid(iDivUp(width, blockW), iDivUp(height, blockH));
    HeatMapArray_gpu << < grid, threadBlock >>
        > (pix_count, dev_count, dev_kernel, kernel_size, width, height);

    float *color_count;
    hipMalloc((void **) &color_count, window_size * sizeof(float));
    hipMemset(color_count, 0, window_size * sizeof(float));
    int64_t mean_radius = (int) (log((kernel_size - 3) / 2) / 0.4);

    MeanKernel_gpu << < grid, threadBlock >>
        > (dev_count, color_count, mean_radius + 1, width, height);
    MeanKernel_gpu << < grid, threadBlock >>
        > (color_count, dev_count, mean_radius / 2 + 1, width, height);

    auto host_count = (float *) malloc(window_size * sizeof(float));
    hipMemcpy(host_count, dev_count, window_size * sizeof(float), hipMemcpyDeviceToHost);
    float max_pix = 0;
    for (auto k = 0; k < window_size; k++) {
        if (max_pix < host_count[k])
            max_pix = host_count[k];
    }
    ColorGradient color_gradient;
    color_gradient.createDefaultHeatMapGradient();
    colors_ = (float*) malloc(window_size * 4 * sizeof(float));

    int64_t c_offset = 0;
    for (auto j = 0; j < window_size; j++) {
        float value = host_count[j] / max_pix;
        float color_r, color_g, color_b;
        color_gradient.getColorAtValue(value, color_r, color_g, color_b);
        colors_[c_offset++] = color_r;
        colors_[c_offset++] = color_g;
        colors_[c_offset++] = color_b;
        colors_[c_offset++] = value;
    }
    
    free(kernel);
    free(host_count);
    hipFree(pix_count);
    hipFree(dev_kernel);
    hipFree(dev_count);
    hipFree(color_count);
}

} //namespace render
} //namespace
