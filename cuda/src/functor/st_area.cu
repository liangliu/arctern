#include "hip/hip_runtime.h"
//
// Created by mike on 2/10/20.
//

#include "functor/st_area.h"
#include "common/gpu_memory.h"

namespace zilliz {
namespace gis {
namespace cpp {

inline DEVICE_RUNNABLE double
ST_area_polygon(const GeoContext& ctx, int index) {
    auto meta = ctx.get_meta_ptr(index);
    auto value = ctx.get_value_ptr(index);
    assert(meta[0] == 1);
    auto count = (int)meta[1];
    assert(count == 5);
    double sum_area = 0;
    for (int point_index = 0; point_index < count; ++point_index) {
        auto lv = value + 2 * point_index;
        auto rv = (point_index + 1 == count) ? value : lv + 2;
        auto area = lv[0] * rv[1] - lv[1] * rv[0];
        sum_area += area;
    }
    return fabs(sum_area / 2);
}


__global__ void
ST_area_kernel(GeoContext ctx, double* result) {
    auto tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < ctx.size) {
        auto tag = ctx.get_tag(tid);
        // handle 2d case only for now
        assert(tag.get_group() == WKB_Group::None);
        switch (tag.get_category()) {
            case WKB_Category::Polygon: {
                result[tid] = ST_area_polygon(ctx, tid);
                break;
            }
            default: {
                assert(false);
            }
        }
    }
}

void
ST_area(const GeometryVector& vec, double* host_results) {
    auto ctx = vec.create_gpuctx();
    auto config = GetKernelExecConfig(vec.size());
    auto dev_result = gpu_make_unique_array<double>(vec.size());
    ST_area_kernel<<<config.grid_dim, config.block_dim>>>(ctx.get(), dev_result.get());
    gpu_memcpy(host_results, dev_result.get(), vec.size());
}

}    // namespace cpp
}    // namespace gis
}    // namespace zilliz
