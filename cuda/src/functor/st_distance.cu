#include "hip/hip_runtime.h"
//
// Created by mike on 2/10/20.
//
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <cmath>
#include "functor/st_distance.h"
#include "common/gpu_memory.h"

namespace zilliz {
namespace gis {
namespace cpp {

inline DEVICE_RUNNABLE double
ST_distance_point_point(const GeoContext& left, const GeoContext& right, int index) {
    auto lv = left.get_value_ptr(index);
    auto rv = right.get_value_ptr(index);
    auto dx = (lv[0] - rv[0]);
    auto dy = (lv[1] - rv[1]);
    return sqrt(dx * dx + dy * dy);
}

__global__ void
ST_distance_kernel(GeoContext left, GeoContext right, double* result) {
    auto tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < left.size) {
        auto left_tag = left.get_tag(tid);
        auto right_tag = right.get_tag(tid);
        // handle 2d case only for now
        assert(left_tag.get_group() == WkbGroup::None);
        assert(right_tag.get_group() == WkbGroup::None);
        if (left_tag.get_category() == WkbCategory::Point &&
            right_tag.get_category() == WkbCategory::Point) {
            result[tid] = ST_distance_point_point(left, right, tid);
        } else {
            result[tid] = NAN;
        }
    }
}

void
ST_distance(const GeometryVector& left,
            const GeometryVector& right,
            double* host_results) {
    assert(left.size() == right.size());
    auto left_ctx = left.create_gpuctx();
    auto right_ctx = right.create_gpuctx();
    auto config = GetKernelExecConfig(left.size());
    auto dev_result = GpuMakeUniqueArray<double>(left.size());
    ST_distance_kernel<<<config.grid_dim, config.block_dim>>>(
        left_ctx.get(), right_ctx.get(), dev_result.get());
    GpuMemcpy(host_results, dev_result.get(), left.size());
}

}    // namespace cpp
}    // namespace gis
}    // namespace zilliz
