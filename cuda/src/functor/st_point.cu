#include "hip/hip_runtime.h"
#include "functor/st_point.h"

namespace zilliz {
namespace gis {
namespace cuda {

constexpr int max_buffer_per_meta = 50;
constexpr int max_buffer_per_value = 250;
using DataState = GeometryVector::DataState;


struct OutputInfo {
    WkbTag tag;
    int meta_size;
    int value_size;
};

__device__ inline OutputInfo
ST_point_compute_kernel(const double* xs,
                        const double* ys,
                        int index,
                        uint32_t* meta_output,
                        double* value_output,
                        bool skip_write = false) {
    (void)meta_output;
    if (!skip_write) {
        assert(value_output != nullptr);
        value_output[0] = xs[index];
        value_output[1] = ys[index];
    }
    return OutputInfo{WkbTag(WkbCategory::Point, WkbGroup::None), 0, 2};
}

__global__ void
ST_point_reserve_kernel(const double* xs, const double* ys, GeoContext results) {
    assert(results.data_state == DataState::FlatOffset_EmptyInfo);
    auto index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < results.size) {
        auto out_info = ST_point_compute_kernel(xs, ys, index, nullptr, nullptr, true);
        results.tags[index] = out_info.tag;
        results.meta_offsets[index] = out_info.meta_size;
        results.value_offsets[index] = out_info.value_size;
    }
}


DEVICE_RUNNABLE inline void
check_info(OutputInfo info, const GeoContext& ctx, int index) {
    assert(info.tag.data_ == ctx.get_tag(index).data_);
    assert(info.meta_size == ctx.meta_offsets[index + 1] - ctx.meta_offsets[index]);
    assert(info.value_size == ctx.value_offsets[index + 1] - ctx.value_offsets[index]);
}

static
__global__ void
ST_point_datafill_kernel(const double* xs,
                         const double* ys,
                         int size,
                         GeoContext results,
                         GeoWorkspace) {
    assert(results.data_state == DataState::PrefixSumOffset_EmptyData);
    auto index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < results.size) {
        auto meta_output = results.get_meta_ptr(index);
        auto value_output = results.get_value_ptr(index);
        auto out_info = ST_point_compute_kernel(xs, ys, index, meta_output, value_output);
        check_info(out_info, results, index);
    }
}


// GeometryVector
// ST_point(const double* xs, const double ys, size_t size) {
//
//}

}    // namespace cuda
}    // namespace gis
}    // namespace zilliz
