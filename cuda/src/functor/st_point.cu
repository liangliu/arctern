#include "hip/hip_runtime.h"
#include "functor/st_point.h"
#include "common/gpu_memory.h"

namespace zilliz {
namespace gis {
namespace cuda {

using DataState = GeometryVector::DataState;
struct OutputInfo {
    WkbTag tag;
    int meta_size;
    int value_size;
};

__device__ inline OutputInfo
ST_point_calc(const double* xs,
              const double* ys,
              int index,
              GeoContext& results,
              bool skip_write = false) {
    if (!skip_write) {
        auto value = results.get_value_ptr(index);
        value[0] = xs[index];
        value[1] = ys[index];
    }
    return OutputInfo{WkbTag(WkbCategory::Point, WkbGroup::None), 0, 2};
}

__global__ void
ST_point_reserve_kernel(const double* xs, const double* ys, GeoContext results) {
    assert(results.data_state == DataState::FlatOffset_EmptyInfo);
    auto index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < results.size) {
        auto out_info = ST_point_calc(xs, ys, index, results, true);
        results.tags[index] = out_info.tag;
        results.meta_offsets[index] = out_info.meta_size;
        results.value_offsets[index] = out_info.value_size;
    }
}


DEVICE_RUNNABLE inline void
check_info(OutputInfo info, const GeoContext& ctx, int index) {
    assert(info.tag.data_ == ctx.get_tag(index).data_);
    assert(info.meta_size == ctx.meta_offsets[index + 1] - ctx.meta_offsets[index]);
    assert(info.value_size == ctx.value_offsets[index + 1] - ctx.value_offsets[index]);
}

static __global__ void
ST_point_datafill_kernel(const double* xs, const double* ys, GeoContext results) {
    assert(results.data_state == DataState::PrefixSumOffset_EmptyData);
    auto index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < results.size) {
        auto out_info = ST_point_calc(xs, ys, index, results);
        check_info(out_info, results, index);
    }
}


void
ST_point(const double* cpu_xs, const double* cpu_ys, int size, GeometryVector& results) {
    results.OutputInitialize(size);
    auto xs = GpuMakeUniqueArrayAndCopy(cpu_xs, size);
    auto ys = GpuMakeUniqueArrayAndCopy(cpu_ys, size);
    auto ctx = results.OutputCreateGeoContext();
    {
        auto config = GetKernelExecConfig(size);
        ST_point_reserve_kernel<<<config.grid_dim, config.block_dim>>>(
            xs.get(), ys.get(), ctx.get());
        ctx->data_state = DataState::FlatOffset_FullInfo;
    }
    results.OutputEvolveWith(ctx.get());
    {
        auto config = GetKernelExecConfig(size, 1);
        ST_point_datafill_kernel<<<config.grid_dim, config.block_dim>>>(
            xs.get(), ys.get(), ctx.get());
        ctx->data_state = DataState::PrefixSumOffset_FullData;
    }
    results.OutputFinalizeWith(ctx.get());
}

}    // namespace cuda
}    // namespace gis
}    // namespace zilliz
